#include "hip/hip_runtime.h"
/**
* This file is part of sublabel_relax.
*
* Copyright 2016 Thomas Möllenhoff <thomas dot moellenhoff at in dot tum dot de> 
* and Emanuel Laude <emanuel dot laude at in dot tum dot de> (Technical University of Munich)
*
* sublabel_relax is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License, or
* (at your option) any later version.
*
* prost is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with sublabel_relax. If not, see <http://www.gnu.org/licenses/>.
*/

#include "block_dataterm_sublabel.hpp"

namespace prost {

template<typename T>
__global__
void BlockDatatermSublabelKernel(T *d_res,
				 const T *d_rhs,
				 size_t nx,
				 size_t ny,
				 size_t L,
				 T t_min,
				 T t_max)
{
  size_t x = threadIdx.x + blockDim.x * blockIdx.x;
  size_t y_tilde = threadIdx.y + blockDim.y * blockIdx.y;
  size_t l = y_tilde % (L-1);
  size_t y = y_tilde / (L-1);
  
  if(x >= nx || y >= ny || l >= L-1)
    return;
  
  size_t idx = (L-1)*ny*x + y*(L-1) + l;
  
  T delta_t = (t_max - t_min) / (L-1);
  T t = t_min + l * delta_t;
  
  T add_v = -t*d_rhs[idx];
  for(size_t i = 1; i < L-1-l; i++)
  {
    add_v += delta_t*d_rhs[idx+i];
  }

  d_res[idx] += add_v;
}

template<typename T>
__global__
void BlockDatatermSublabelAdjointKernel(T *d_res,
					const T *d_rhs,
					size_t nx,
					size_t ny,
					size_t L,
					T t_min,
					T t_max)
{
  size_t x = threadIdx.x + blockDim.x * blockIdx.x;
  size_t y_tilde = threadIdx.y + blockDim.y * blockIdx.y;
  size_t l = y_tilde % (L-1);
  size_t y = y_tilde / (L-1);

  if(x >= nx || y >= ny || l >= L-1)
    return;

  size_t idx = (L-1)*ny*x + y*(L-1) + l;
  
  T delta_t = (t_max - t_min) / (L-1);
  T t = t_min + l * delta_t;
  
  T add_s = -t*d_rhs[idx];
  for(size_t i = 1; i <= l; i++)
    add_s += delta_t*d_rhs[idx-i];

  d_res[idx] += add_s;
}
  
template<typename T>
BlockDatatermSublabel<T>::BlockDatatermSublabel(size_t row, 
						size_t col, 
						size_t nx, 
						size_t ny, 
						size_t L, 
						T left, 
						T right)
  : Block<T>(row,col,nx*ny*(L-1),nx*ny*(L-1)), nx_(nx), ny_(ny), L_(L), t_min_(left), t_max_(right)
{
}

template<typename T>
T BlockDatatermSublabel<T>::row_sum(size_t row, T alpha) const
{
  size_t l = row % (L_-1);
  T delta_t = (t_max_ - t_min_) / (L_-1);
  T t = t_min_ + l * delta_t;
  return t + (L_-2-l)*delta_t;  
}

template<typename T>
T BlockDatatermSublabel<T>::col_sum(size_t col, T alpha) const
{
  size_t l = col % (L_-1);
  T delta_t = (t_max_ - t_min_) / (L_-1);
  T t = t_min_ + l * delta_t;

  return t + l*delta_t;
}

template<typename T>
void BlockDatatermSublabel<T>::EvalLocalAdd(
    const typename device_vector<T>::iterator& res_begin,
    const typename device_vector<T>::iterator& res_end,
    const typename device_vector<T>::const_iterator& rhs_begin,
    const typename device_vector<T>::const_iterator& rhs_end)
{
  dim3 block(1, 128, 1);
  dim3 grid((nx_ + block.x - 1) / block.x,
            (ny_*(L_-1) + block.y - 1) / block.y,
            1);

  BlockDatatermSublabelKernel<T>
    <<<grid, block>>>(thrust::raw_pointer_cast(&(*res_begin)),
		      thrust::raw_pointer_cast(&(*rhs_begin)),
		      nx_,
		      ny_,
		      L_,
		      t_min_,
		      t_max_);
}

template<typename T>
void BlockDatatermSublabel<T>::EvalAdjointLocalAdd(
    const typename device_vector<T>::iterator& res_begin,
    const typename device_vector<T>::iterator& res_end,
    const typename device_vector<T>::const_iterator& rhs_begin,
    const typename device_vector<T>::const_iterator& rhs_end)
{
  dim3 block(1, 128, 1);
  dim3 grid((nx_ + block.x - 1) / block.x,
            (ny_*(L_-1) + block.y - 1) / block.y,
            1);

  BlockDatatermSublabelAdjointKernel<T>
    <<<grid, block>>>(thrust::raw_pointer_cast(&(*res_begin)),
		      thrust::raw_pointer_cast(&(*rhs_begin)),
		      nx_,
		      ny_,
		      L_,
		      t_min_,
		      t_max_);
}

// Explicit template instantiation
template class BlockDatatermSublabel<float>;
template class BlockDatatermSublabel<double>;

} // namespace prost